#include "hip/hip_runtime.h"
/*+++++++++++++++++++++++++++++++++

Project: ParaDIS (Parallel Algorithm for Imputation of Missing Values in Streaming Time Series)

Source file: imputeValue.cu

Purpose: Imputation one missing point in time series 

Author(s): Andrey Poluyanov (andrey.poluyanov@gmail.com) and Mikhail Zymbler (mzym@susu.ru)

+++++++++++++++++++++++++++++++++*/

#include "imputeValue.cuh"

itemType Q[d_par][l_par];
itemType SetC[d_par][subseq_count + numimputevalue][l_par];
itemType LB[d_par][subseq_count + numimputevalue][num_lb];
itemType bsf[d_par];
itemType N[d_par][subseq_count + numimputevalue];
int CandIndex[subseq_count + numimputevalue];
int ind[k_par];
itemType RANK[subseq_count + numimputevalue];
bool BitMap[d_par][subseq_count + numimputevalue];
extern int num_of_threads;
extern int DTW_count;
extern int LB_count;

struct maxRank{
	itemType Value;
	int Indax;
};
#pragma omp declare reduction(max : struct maxRank : \
	omp_out.Value = omp_in.Value < omp_out.Value ? omp_out.Value : omp_in.Value, \
	omp_out.Indax = omp_in.Value < omp_out.Value ? omp_out.Indax : omp_in.Indax ) \
	initializer( omp_priv = { 0, 0 } )

#pragma omp declare reduction(min : struct maxRank : \
	omp_out.Value = omp_in.Value > omp_out.Value ? omp_out.Value : omp_in.Value, \
	omp_out.Indax = omp_in.Value > omp_out.Value ? omp_out.Indax : omp_in.Indax ) \
	initializer( omp_priv = { inf_val, 0 } )



itemType impute(itemType* S, itemType** R, int h, itemType *SetC_gpu, itemType *Q_gpu, itemType *LB_gpu)
{

    itemType s = 0;
	PRF_START(start2);
	FillData(S, R, h);
	PRF_FINISH(finish2);
	PRF_START(start3);
	CslcZnorm(h);
	PRF_FINISH(finish3);
	CalcLB(h, SetC_gpu, Q_gpu, LB_gpu);
	
	PRF_START(start7);
	verticalOverlap(h, use_corr);
	int i, j = 0;
	int rank_counts = 0;
	while (j < k_par)
	{
		struct maxRank maxRANK = {0, 0};
		#pragma omp parallel for num_threads(num_of_threads) reduction (max: maxRANK)
		for (i = 0; i <= subseq_count + h; i++)
		{
			if (maxRANK.Value < RANK[i])
			{ 
				maxRANK.Value = RANK[i];
				maxRANK.Indax = i;
			}
		}

		if(maxRANK.Value>0)
		{
			ind[j] = maxRANK.Indax;
			int t;
			for (t = max(0, (ind[j] - l_par)); t < min((ind[j] + l_par),(subseq_count + h + 1)); t++)
			{
				RANK[t] = 0;
			}
			rank_counts+=1;
		}
		else break;
		j++;
	}

	assert(rank_counts>0);

	#ifdef DEBUG
		DTW_count += d_par;
		LB_count += d_par*(subseq_count + h + 1);	
		for (j = 0; j < d_par; j++)
		{
			for (int i = 0; i <= subseq_count + h; i++)
			{
				if (N[j][i] < inf_val)
				{
					DTW_count++;
				}
			}
		}
	#endif

	int n = 0;
	for (j = 0; j < k_par; j++) {
		if (ind[j] != -1) 
		{
			s = s + S[ind[j] + l_par -1];
			n++;
		}
	}
	assert(n>0);
	
	PRF_FINISH(finish7);
	return s/n;
}

void FillData(itemType* S, itemType** R, int h)
{
	int i, j, q;
	for (j = 0; j < d_par; j++)
	{
		for (i = 0; i < l_par; i++)
			Q[j][i] = R[j][L_par - l_par + h + 1 + i];
	}
	for (i = 0; i < d_par; i++)
	{
		for (j = 0; j <= subseq_count + h; j++)
		{
			for (q = 0; q < l_par; q++)
				SetC[i][j][q] = R[i][j + q];
		}
	}
	for (i = 0; i < d_par; i++)
	{
		for (j = 0; j <= subseq_count + h; j++)
			N[i][j] = inf_val;
	}

	for (i = 0; i < k_par; i++)
		ind[i] = -1;
	
	for (j = 0; j <= subseq_count + h; j++)
	{
		RANK[j] = 0;
	}

	for (i = 0; i < d_par; i++)
	{
		for (j = 0; j <= subseq_count + h; j++)
			BitMap[i][j] = true;
	}
}

void CslcZnorm(int h)
{
	int i ,j;
	#pragma omp parallel for num_threads(num_of_threads) collapse(2) 
	for (i = 0; i < d_par; i++)
	{
		for (j = 0; j <= subseq_count + h; j++) 
			Z_norm(SetC[i][j], l_par);
	}
	#pragma omp parallel for num_threads(num_of_threads)
	for (i = 0; i < d_par; i++)
	{
		Z_norm(Q[i], l_par);
	}
}

int CalcLB(int h, itemType *SetC_gpu, itemType *Q_gpu, itemType *LB_gpu)
{
	int i,j,s,cnt,left,right;
	itemType cur_dist;
	PRF_START(start4);
	
	
	if (use_GPU==true)
	{
		int N_subs = subseq_count + numimputevalue;
		
		hipMemcpy(SetC_gpu, SetC, d_par*(subseq_count + numimputevalue)*l_par*sizeof(itemType), hipMemcpyHostToDevice);
		hipMemcpy(Q_gpu, Q, d_par*l_par*sizeof(itemType), hipMemcpyHostToDevice);
		
		dim3 grid, block;
		int blockSize = 512;
		
		grid.x = (N_subs + blockSize - 1) / blockSize;  grid.y = d_par;
		block.x = blockSize; block.y = 1;
		
		LB_calc_kernel<<<grid, block>>>(Q_gpu, SetC_gpu, LB_gpu, N_subs, (subseq_count + h + 1)); 
		hipMemcpy(LB, LB_gpu, d_par*(subseq_count + numimputevalue)*num_lb*sizeof(itemType), hipMemcpyDeviceToHost);
	}
	else
	{		
	
		//CPU
		#pragma omp parallel for num_threads(num_of_threads) collapse(2)
		for (i = 0; i < d_par; i++)
		{
			for (j = 0; j <= subseq_count + h; j++)
			{
				LB[i][j][0] = LbKim(Q[i], SetC[i][j]);
				LB[i][j][1] = LbKeogh(Q[i], SetC[i][j], r_par);
				LB[i][j][2] = LbKeogh(SetC[i][j], Q[i], r_par);
				LB[i][j][3] = max(LB[i][j][1], LB[i][j][2]);
			}
		}
	
	}
	
	PRF_FINISH(finish4);


	for (int i = 0; i < d_par; i++)
	{
		bsf[i] = inf_val;
		if (init_bsf==0)
		{
			bsf[i] = DTW(Q[i], SetC[i][(L_par - (3 * l_par) + h)], r_par) + itemtype_epsilon;
		}
		else if (init_bsf==1)
		{
			bsf[i] = DTW(Q[i], SetC[i][subseq_count + h], r_par) + itemtype_epsilon;
		}
		else if (init_bsf==2)
		{
			struct maxRank minLB = {inf_val, 0};
			#pragma omp parallel for num_threads(num_of_threads) reduction (min: minLB)
			for (j = 0; j <= subseq_count + h; j++)
			{
				if (minLB.Value > LB[i][j][3])
				{ 
					minLB.Value = LB[i][j][3];
					minLB.Indax = j;
				}
			}
			bsf[i] = DTW(Q[i], SetC[i][minLB.Indax], r_par) + itemtype_epsilon;
		}
	}
		
	for (i = 0; i < d_par; i++)
	{
		while(true)
		{
			PRF_START(start5);
			#pragma omp parallel for num_threads(num_of_threads)
			for (j = 0; j <= subseq_count + h; j++)
			{
				BitMap[i][j] = BitMap[i][j] && (bsf[i] > LB[i][j][0]) && (bsf[i] > LB[i][j][3]);
			}
			cnt = 0;

			if(cut_off_neighbors==true)
			{
				for (j = 0; j <= subseq_count + h; j++)
				{
					if (BitMap[i][j])
					{
						if ((cnt > 0) && ((j - CandIndex[cnt-1]) < l_par))
						{
 							if (LB[i][j][3] < LB[i][CandIndex[cnt-1]][3])
							{
 								CandIndex[cnt-1] = j;
							} 
						}
						else
						{
							CandIndex[cnt] = j;
							cnt++;
						}
					}
				}
			}
			else
			{			
				for (j = 0; j <= subseq_count + h; j++)
				{
					if (BitMap[i][j])
					{
						CandIndex[cnt] = j;
						cnt++;
					}
				}
			}
			
			PRF_FINISH(finish5);
			if (cnt == 0) break;

			PRF_START(start6);
			s = 1;
			while(true)
			{
				left = num_of_threads*(s-1);
				right = min(cnt, num_of_threads*s);
				cur_dist = bsf[i];
				
				#pragma omp parallel for num_threads(num_of_threads)
				for (j = left; j < right; j++)
				{
					N[i][CandIndex[j]] = DTW(Q[i], SetC[i][CandIndex[j]], r_par);
					BitMap[i][CandIndex[j]] = false;
				}
				
				for (j = left; j < right; j++)
					if (cur_dist > N[i][CandIndex[j]]) cur_dist = N[i][CandIndex[j]];
								
				s+=1;
				if ((right==cnt)||(bsf[i]>cur_dist)) 
				{
					bsf[i] = min(bsf[i], cur_dist);
					break;
				}
			}
			PRF_FINISH(finish6);
		}
	}
	return 0;
}

void verticalOverlap(int h, bool corr)
{
	int i ,j;
		
	for (j = 0; j < d_par; j++)
	{
		#pragma omp parallel for num_threads(num_of_threads)
		for (i = 0; i <= subseq_count + h; i++)
		{
			if (N[j][i] < inf_val)
			{
				if(corr == true)
				{
					RANK[i] = RANK[i] + ((1 / (N[j][i]+itemtype_epsilon)) * ((d_par - j) / d_par));
				}
				else
				{
					RANK[i] = RANK[i] + (1 / (N[j][i]+itemtype_epsilon));
				}
			}
		}
	}
}


__host__ __device__ itemType LbKim(itemType* Q, itemType* C)
{
	return (dist(Q[0], C[0]) + dist(Q[l_par-1], C[l_par-1]));
}

__host__ __device__ itemType LbKeogh(itemType* Q, itemType* C, int r)
{
	itemType lb_dist = 0;
	itemType min_lokal, max_lokal;

	for (int i = 0; i < l_par; i++)
	{ 
		max_lokal = Q[i];
		min_lokal = Q[i];
		for (int j = max((i - r),0); j <= min((i + r),(l_par-1)); j++)
		{
			max_lokal = max(max_lokal, Q[j]);
			min_lokal = min(min_lokal, Q[j]);
		}
			
		if (C[i] > max_lokal)
		{
			lb_dist = lb_dist + dist(C[i], max_lokal);
		}
		else if (C[i] < min_lokal)
		{
			lb_dist = lb_dist + dist(C[i], min_lokal);
		}
	}
	return lb_dist;
}   

__global__ void LB_calc_kernel(itemType *Q, itemType *SetC, itemType *LB, int N, int idx_max)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
 
 	if(idx<idx_max)
    {

        int s_pos_y=idy*l_par*N;
        int s_pos_x=idx*l_par;
        int start = idy*num_lb*N + idx*num_lb;      
        
        LB[start+0] = LbKim(&Q[idy*l_par],&SetC[s_pos_y + s_pos_x]);
        LB[start+1] = LbKeogh(&Q[idy*l_par],&SetC[s_pos_y + s_pos_x],r_par);
        LB[start+2] = LbKeogh(&SetC[s_pos_y + s_pos_x],&Q[idy*l_par],r_par);
        LB[start+3] = max(LB[start+1], LB[start+2]);
    }
}

